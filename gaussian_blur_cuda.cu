

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define PGM_ID_LINE "P5"
#define PGM_ID_LINE_LEN 3
#define PI 3.14159265358979323846
#define NSEC_PER_SEC 1000000000

#define idx(arr, cols, i, j) (arr[(cols)*(i) + (j)])


struct compute_data {
    unsigned char *in_image_h;
    unsigned char *out_image_h;
    float *gaussian_h;
    int width;
    int height;
    int max_pixel_value;
    float sigma;
    int order;
};


/* Offset from a Gaussian's center for x, y at index 0, 0.
 *
 * For example, for order 5, -2 is returned.
 */
__host__ __device__ static inline float goffset(float order)
{
    return -(order - 1)/2;
}

__host__ __device__ static inline int clamp(int x, int low, int high)
{
    if (x < low) x = low;
    if (x > high) x = high;
    return x;
}

#define cuda_check(ret) _cuda_check((ret), __FILE__, __LINE__)
inline void _cuda_check(hipError_t ret, const char *file, int line)
{
    if (ret != hipSuccess) {
        fprintf(stderr, "CudaErr: %s (%s:%d)\n", hipGetErrorString(ret), file, line);
        exit(1);
    }
}

__global__ void convolution_kernel(unsigned char *in_d, unsigned char *out_d, float *gauss_d, int width, int height, int order){
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    float term = 0;
    int xsub, ysub;
    int offset = goffset(order);
    int i, j;

    if (x >= width || y >= height)
        return;

    for (i = 0; i < order; i++) {
        ysub = clamp(y + offset + i, 0, height - 1);

        for (j = 0; j < order; j++) {
            xsub = clamp(x + offset + j, 0, width - 1);

            term += idx(in_d, width, ysub, xsub)
                * idx(gauss_d, order, i, j);
        }
    }

    idx(out_d, width, y, x) = term;
}


static int get_max_threads()
{
    int max_threads;
    struct hipDeviceProp_t properties;

    hipGetDeviceProperties(&properties, 0);
    max_threads = properties.maxThreadsPerBlock;

    return max_threads;
}


/* Initialize the Gaussian convolution matrix.
 */
static void init_gaussian(struct compute_data *data)
{
    float sum = 0;
    float x, y, res;
    float offset = goffset(data->order);
    int i, j;

    for (i = 0, y = offset; i < data->order; i++, y++){
        for (j = 0, x = offset; j < data->order; j++, x++){
            res = exp(-(x*x + y*y)/(2*data->sigma*data->sigma));
            //res /= 2*PI*data->sigma*data->sigma;

            sum += res;
            idx(data->gaussian_h, data->order, i, j) = res;
        }
    }

    for (int i = 0; i < data->order; i++){
        for (int j = 0; j < data->order; j++){
            idx(data->gaussian_h, data->order, i, j) /= sum;
        }
    }
}


/* Convolute the Gaussian with the input image.
 */
static void apply_gaussian(struct compute_data *data)
{
    unsigned char *in_d, *out_d;
    float *gauss_d;
    int gx, gy, bx, by;
    int size = data->width * data->height * sizeof(*in_d); // size for in and out
    int gauss_size = data->order * data->order * sizeof(*gauss_d);

    // Load data
    cuda_check(hipMalloc(&in_d, size));
    cuda_check(hipMalloc(&out_d,size));
    cuda_check(hipMalloc(&gauss_d,gauss_size));

    cuda_check(hipMemcpy(in_d, data->in_image_h, size, hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(gauss_d, data->gaussian_h, gauss_size, hipMemcpyHostToDevice));

    /* Compute grid and block sizes.
     * A = number of threads (area)
     * w' = s*w    h' = s*h
     * w'*h' = A
     * s^2*w*h = A
     * s = sqrt(A/(w*h))
     */
    bx = data->width * sqrt(get_max_threads()) / sqrt(data->width * data->height);
    by = data->height * sqrt(get_max_threads()) / sqrt(data->width * data->height);

    gx = data->width/bx + 1;
    gy = data->height/by + 1;

#ifdef DEBUG
    printf("grid(%d, %d), block(%d, %d)\n", gx, gy, bx, by);
#endif

    // Run kernel
    dim3 grid_dim(gx, gy);
    dim3 block_dim(bx, by);
    convolution_kernel<<<grid_dim, block_dim>>>(in_d, out_d, gauss_d,
            data->width, data->height, data->order);

    cuda_check(hipPeekAtLastError());
    cuda_check(hipDeviceSynchronize());

    // Copy back to host
    cuda_check(hipMemcpy(data->out_image_h, out_d, size, hipMemcpyDeviceToHost));

    cuda_check(hipFree(in_d));
    cuda_check(hipFree(out_d));
    cuda_check(hipFree(gauss_d));
}


// Return time passed in seconds.
static float get_timespec_delta(const struct timespec *start,
        const struct timespec *stop)
{
    long long delta_nsec, start_nsec, stop_nsec;

    start_nsec = start->tv_sec * NSEC_PER_SEC + start->tv_nsec;
    stop_nsec = stop->tv_sec * NSEC_PER_SEC + stop->tv_nsec;
    delta_nsec = stop_nsec - start_nsec;

    return (float)delta_nsec / NSEC_PER_SEC;
}


int main(int argc, char *argv[])
{
    FILE *in_file;
    FILE *out_file;
    int amount, amount_read;
    struct compute_data data;
    struct timespec start, stop;

    if (argc < 4) {
        fprintf(stderr, "Usage: %s <input_file> <output_file> <sigma>\n", argv[0]);
        return 1;
    }

    in_file = fopen(argv[1], "r");
    if (NULL == in_file) {
        fprintf(stderr, "Error: cannot open file %s\n", argv[1]);
        return 1;
    }

    // Read pgm metadata.
    char id_line[PGM_ID_LINE_LEN + 1];
    if (NULL == fgets(id_line, PGM_ID_LINE_LEN, in_file)
            || strcmp(id_line, PGM_ID_LINE) != 0) {
        fprintf(stderr, "Error: invalid PGM information\n");
        return 1;
    }

    if (fscanf(in_file, "%d %d\n%d\n", &data.width, &data.height,
                &data.max_pixel_value) < 3) {
        fprintf(stderr, "Error: invalid PGM information\n");
        return 1;
    }

    // Read image data.
    amount = data.width*data.height;
    data.in_image_h = (unsigned char*) malloc(amount*sizeof(*data.in_image_h));
    amount_read = fread(data.in_image_h, sizeof(*data.in_image_h), amount, in_file);

    if (amount_read < amount) {
        fprintf(stderr, "Error: invalid PGM pixels\n");
        return 1;
    }

    fclose(in_file);


    // Determine sigma and order.
    char *end;
    data.sigma = strtod(argv[3], &end);
    if (end == argv[3] || data.sigma <= 0) {
        fprintf(stderr, "Error: invalid sigma value\n");
        return 1;
    }

    data.order = ceil(6*data.sigma);
    if (data.order % 2 == 0)
        data.order += 1;
    if (data.order > data.width || data.order > data.height) {
        fprintf(stderr, "Error: sigma value too big for image size\n");
        return 1;
    }


    // Compute image
    data.out_image_h = (unsigned char *) malloc(amount*sizeof(*data.out_image_h));
    data.gaussian_h = (float *) malloc(data.order*data.order*sizeof(*data.gaussian_h));


    clock_gettime(CLOCK_MONOTONIC, &start);

    init_gaussian(&data);

#ifdef DEBUG
    for (int i = 0; i < data.order; i++) {
        for (int j = 0; j < data.order; j++) {
            printf("%f ", idx(data.gaussian_h, data.order, i, j));
        }
        printf("\n");
    }
#endif

    apply_gaussian(&data);

    clock_gettime(CLOCK_MONOTONIC, &stop);
    printf("Running time: %.6f secs\n", get_timespec_delta(&start, &stop));


    // Output_image
    out_file = fopen(argv[2], "w");
    fprintf(out_file, "%s\n%d %d\n%d\n", PGM_ID_LINE, data.width, data.height,
            data.max_pixel_value);
    fwrite(data.out_image_h, sizeof(char), amount, out_file);
    fclose(out_file);


    free(data.in_image_h);
    free(data.out_image_h);

    return 0;
}
